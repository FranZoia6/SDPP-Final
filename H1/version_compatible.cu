#include <stdio.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line) {
	if (code != hipSuccess) {
    	fprintf(stderr, "GPUassert: %s en %s en la línea %d\n",
                     hipGetErrorString(code), file, line);
    	exit(code);
	}
}

int main() {
    gpuErrchk(hipDeviceSynchronize());
    return 0;
}
