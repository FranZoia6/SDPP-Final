global void holaMundo() {
    printf("Hola Mundo desde el GPU!\n");
}

int main() {
    // Lanzar el kernel
    holaMundo<<<1, 10>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize(); // Esperar a que el GPU termine
    return 0;
} 